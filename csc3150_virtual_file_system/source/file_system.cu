﻿
#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ __managed__ u32 gtime = 0;


__device__ void fs_init(FileSystem *fs, uchar *volume, int SUPERBLOCK_SIZE,
							int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
							int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE, 
							int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS)
{
  // init variables
  fs->volume = volume;

  // init constants
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;
  fs->FCB_SIZE = FCB_SIZE;
  fs->FCB_ENTRIES = FCB_ENTRIES;
  fs->STORAGE_SIZE = VOLUME_SIZE;
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;
  fs->MAX_FILE_NUM = MAX_FILE_NUM;
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;
  fs->VCB = reinterpret_cast<VCB *> (fs->volume);
  fs->FCB = reinterpret_cast<FCB *> (fs->volume + 50);
  fs->SB = reinterpret_cast<uchar *> (fs->volume + fs->FILE_BASE_ADDRESS);
}

__device__ bool cmp(char *A, char *B){
  for(int i = 0; i < 20; i++)
    if(A[i] != B[i]) return 0;
    else if(A[i] == '\0' && B[i] == '\0') return 1;
    return 0;
}
__device__ void cpy(char *A, char *B){
  for(int i = 0; i < 20; i++){
    A[i] = B[i];
    if(B[i] == '\0') return;
  }
}
__device__ u32 fs_open(FileSystem *fs, char *s, int op)
{
	/* Implement open operation here */
  u32 tot_file_num = fs->VCB->tot_file_num;
  for(int i = 0; i < tot_file_num; i++){
    if(cmp(fs->FCB[i].file_name, s))
      return i;
  }
  if(op == G_WRITE){
    //update vcb
    int fp = fs->VCB->tot_file_num++;
    //update fcb
    fs->FCB[fp].create = gtime;
    fs->FCB[fp].modified = gtime++;
    fs->FCB[fp].size = 0;
    fs->FCB[fp].st_block = fs->VCB->empty_st++;
    cpy(fs->FCB[fp].file_name, s);
    return fp;
  }
  printf("Read file DNE\n");
  return -1;
}


__device__ void fs_read(FileSystem *fs, uchar *output, u32 size, u32 fp)
{
	/* Implement read operation here */
  int st_addr = fs->FCB[fp].st_block * fs->STORAGE_BLOCK_SIZE;
  for(int i = st_addr; i < st_addr + size; i++){
    output[i - st_addr] = fs->SB[i];
  }
}

__device__ u32 fs_write(FileSystem *fs, uchar* input, u32 size, u32 fp)
{
	/* Implement write operation here */
  int block_num = (size + fs->STORAGE_BLOCK_SIZE  - 1) / fs->STORAGE_BLOCK_SIZE;
  int pre_block_num = fs->FCB[fp].size == 0 ? 1 : (fs->FCB[fp].size + fs->STORAGE_BLOCK_SIZE  - 1) / fs->STORAGE_BLOCK_SIZE;
  // overwrite, count previous block;
  int change_num = (block_num - pre_block_num);
  // printf("Write: %s %d %d %d\n", fs->FCB[fp].file_name, change_num, size, pre_block_num);

  // make space for sb
  if(fp < fs->VCB->tot_file_num - 1)
  memcpy(fs->SB + (fs->FCB[fp + 1].st_block + change_num) * fs->STORAGE_BLOCK_SIZE,
  fs->SB + fs->FCB[fp + 1].st_block * fs->STORAGE_BLOCK_SIZE,  (fs->VCB->empty_st - fs->FCB[fp + 1].st_block) * fs->STORAGE_BLOCK_SIZE);
  //update sb
  int st_addr = fs->FCB[fp].st_block * fs->STORAGE_BLOCK_SIZE;
  for(int i = st_addr; i < st_addr + size; i++){
    fs->SB[i] = input[i - st_addr];
  }
  // update vcb
  fs->VCB->empty_st += change_num;
  //update fcb
  fs->FCB[fp].modified = gtime++;
  fs->FCB[fp].size = size;
  for(int i = fp + 1; i < fs->VCB->tot_file_num; i++){
    fs->FCB[i].st_block += change_num;
  }

  return 0; 
}
__device__ void fs_gsys(FileSystem *fs, int op){
	/* Implement LS_D and LS_S operation here */
  if(op == LS_D) {
    printf("===sort by modified time===\n");

    // version 1: use priority to list
      for(u16 i = 0; i < fs->VCB->tot_file_num; i++)
      fs->FCB[i].priority = 2024;
    for(int i = 0; i < fs->VCB->tot_file_num; i++) {
      int idx = -1;
      for(int j = 0; j < fs->VCB->tot_file_num; j++) {
        if(fs->FCB[j].priority == 2024 && (idx == -1 || fs->FCB[idx].modified < fs->FCB[j].modified))
          idx = j;
      }
      fs->FCB[idx].priority = i;
    }
      
    for(int j = 0; j < fs->VCB->tot_file_num; j++)
      for(int i = 0; i < fs->VCB->tot_file_num; i++) {
        if(fs->FCB[i].priority == j)
          printf("%s\n", fs->FCB[i].file_name);
      }
  }
  else if(op == LS_S) {
    printf("===sort by file size===\n");

    // version 1
    for(u16 i = 0; i < fs->VCB->tot_file_num; i++)
      fs->FCB[i].priority = 2024;
    for(int i = 0; i < fs->VCB->tot_file_num; i++) {
      int idx = -1;
      for(int j = 0; j < fs->VCB->tot_file_num; j++) {
        if(fs->FCB[j].priority == 2024 && (idx == -1 || fs->FCB[idx].size < fs->FCB[j].size || 
        (fs->FCB[idx].size == fs->FCB[j].size && fs->FCB[idx].create > fs->FCB[j].create)))
          idx = j;
      }
      fs->FCB[idx].priority = i;
    }
      
    for(int j = 0; j < fs->VCB->tot_file_num; j++)
      for(int i = 0; i < fs->VCB->tot_file_num; i++) {
        if(fs->FCB[i].priority == j)
          printf("%s %u\n", fs->FCB[i].file_name, fs->FCB[i].size);
      }
    // dump(fs);
  } else 
    printf("Wrong operation\n");
  return ;
}

__device__ void fs_gsys(FileSystem *fs, int op, char *s)
{
	/* Implement rm operation here */
  if(op == RM) {
    for(int i = 0; i < fs->VCB->tot_file_num; i++)
      if(cmp(fs->FCB[i].file_name, s)){
          int st_block = fs->FCB[i].st_block;
          int block_num = fs->FCB[i].size == 0 ? 1 : (fs->FCB[i].size + fs->STORAGE_BLOCK_SIZE - 1) / fs->STORAGE_BLOCK_SIZE;
          fs->VCB->tot_file_num--;
          fs->VCB->empty_st -= block_num;
          for(int j = i; j < fs->VCB->tot_file_num; j++){
            fs->FCB[j] = fs->FCB[j + 1];
            fs->FCB[j].st_block -= block_num;
          }
          // compact: move leftover to front
          if(i < fs->VCB->tot_file_num)
            memcpy(fs->SB + st_block * fs->STORAGE_BLOCK_SIZE, fs->SB + (st_block + block_num) * 
          fs->STORAGE_BLOCK_SIZE, (fs->VCB->empty_st - st_block) * fs->STORAGE_BLOCK_SIZE);
          else 
            memset(fs->SB + st_block * fs->STORAGE_BLOCK_SIZE, 0, fs->FCB[i].size);
          return;
        }
    printf("Delete file DNE\n");
  
  } else {
    printf("Wrong operation\n");

  }
}
__device__ void dump(FileSystem *fs){
  for(int i = 0; i < fs->VCB->tot_file_num; i++){
    printf("%d %d %s %d %d %d %d\n", i, fs->FCB[i].st_block, fs->FCB[i].file_name, fs->FCB[i].size, fs->FCB[i].create, fs->FCB[i].modified, fs->FCB[i].priority);
  }
  printf(" empty_st: %d\n", fs->VCB->empty_st);
}
