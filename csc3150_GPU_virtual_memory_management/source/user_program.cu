﻿#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
__device__ void user_program(VirtualMemory *vm, uchar *input, uchar *results,
                             int input_size) {
  for (int i = 0; i < input_size; i++) 
        vm_write(vm, i, input[i]);
	printf("write page fault: %d\n",*(vm->pagefault_num_ptr));
  for (int i = input_size - 1; i >= input_size - 32769; i--) {
      int value = vm_read(vm, i);
  }
	printf("read page fault: %d\n", *(vm->pagefault_num_ptr));

	vm_snapshot(vm, results, 0, input_size);
	printf("snapshot page fault: %d\n", *(vm->pagefault_num_ptr));

  // for(int i = 0; i < input_size; i++)
  //       vm_write(vm, 32*1024+i, input[i]);
	// printf("write page fault: %d\n",*(vm->pagefault_num_ptr));
    
  //   for(int i = 0; i < 32*1023; i++)
  //       vm_write(vm, i, input[i+32*1024]);
	// printf("write page fault: %d\n",*(vm->pagefault_num_ptr));
    
  //   vm_snapshot(vm,results,32*1024,input_size);
	// printf("snapshot page fault: %d\n", *(vm->pagefault_num_ptr));
}
