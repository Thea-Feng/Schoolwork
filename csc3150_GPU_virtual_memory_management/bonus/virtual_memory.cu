﻿#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
__device__ void init_invert_page_table(VirtualMemory *vm) {

  for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
    vm->invert_page_table[i] = 0x80000000; // invalid := MSB is 1
    vm->invert_page_table[i + vm->PAGE_ENTRIES] = i;
  }
}


__device__ void vm_init(VirtualMemory *vm, uchar *buffer, uchar *storage,
                        u32 *invert_page_table, int *pagefault_num_ptr,
                        int PAGESIZE, int INVERT_PAGE_TABLE_SIZE,
                        int PHYSICAL_MEM_SIZE, int STORAGE_SIZE,
                        int PAGE_ENTRIES, u32 *swap_table, URL_node *pool, int thread_id) {
  // init variables
  vm->buffer = buffer;
  vm->storage = storage;
  vm->invert_page_table = invert_page_table;
  vm->pagefault_num_ptr = pagefault_num_ptr;
  vm->swap_table = swap_table;
  vm->pool = pool;
  vm->thread_id = thread_id;

  // init constants
  vm->PAGESIZE = PAGESIZE;
  vm->INVERT_PAGE_TABLE_SIZE = INVERT_PAGE_TABLE_SIZE;
  vm->PHYSICAL_MEM_SIZE = PHYSICAL_MEM_SIZE;
  vm->STORAGE_SIZE = STORAGE_SIZE;
  vm->PAGE_ENTRIES = PAGE_ENTRIES;

  // before first vm_write or vm_read
  init_invert_page_table(vm);

  struct URL_node *head, *now;
  head = (struct URL_node *) malloc(100);
  vm->tail = vm->pool;
  now = vm->tail;
  head->nxt = now;

  for(int i = 0; i < PAGE_ENTRIES; i++){
    now->idx = i;
    if(i != PAGE_ENTRIES - 1){
      now->nxt = ++(vm->tail);
      now = now->nxt;
    } else now->nxt = NULL;
  }
  vm->head = head;

  vm->disk_nxt_available = 0;
  for(int i = 0; i < 5120; i++) vm->swap_table[i] = 0x80000000;

  // struct URL_node 
}
__device__ u32 swap_in_page(VirtualMemory *vm, int pn){
  u32 location = vm->swap_table[pn];
  if(location == 0x80000000) {
    return vm->disk_nxt_available++;
  }
  vm->swap_table[pn] = 0x80000000;
  
  return location;
}
__device__ void swap_out_page(VirtualMemory *vm, int pn, u32 addr){
  if(vm->swap_table[pn] != 0x80000000) {
    printf("ERROR: the element already in disk\n");
    return;
  }
  vm->swap_table[pn] = addr;

  return;
}
__device__ u32 lookup_pt(VirtualMemory *vm, int pn){
  for(int i = 0; i < vm->PAGE_ENTRIES; i++)
    if(vm->invert_page_table[i + vm->PAGE_ENTRIES] == pn && vm->invert_page_table[i] != 0x80000000)
      return vm->invert_page_table[i];
  return 0x80000000;
}

__device__ int LRU_get(VirtualMemory *vm){
  int ret = vm->head->nxt->idx;
  URL_node *tmp = vm->head->nxt;
  vm->head->nxt = vm->head->nxt->nxt;
  vm->tail->nxt = tmp;
  vm->tail = tmp;
  return ret;
}
// __device__ void LRU_update(int idx){

// }
__device__ uchar vm_read(VirtualMemory *vm, u32 addr) {
  /* Complate vm_read function to read single element from data buffer */
  int pn = addr >> 5, offset = addr & 31;
  if(pn % 4 != vm->thread_id) return -1;

  int ret = lookup_pt(vm, pn);
  if(ret == 0x80000000){ //not found
     (*(vm->pagefault_num_ptr))++;

    int idx = LRU_get(vm); // idx of PT
    u32 replace_page = vm->invert_page_table[idx]; // frame addr

    u32 pos_in = swap_in_page(vm, pn);

    for(int i = 0; i < 32; i++) {
        uchar tmp = vm->buffer[(replace_page << 5) + i];
        vm->buffer[(replace_page << 5) + i] = vm->storage[(pos_in << 5) + i];
        vm->storage[(pos_in << 5) + i] = tmp;
      }
    
    swap_out_page(vm, vm->invert_page_table[idx + vm->PAGE_ENTRIES], pos_in);
    vm->invert_page_table[idx + vm->PAGE_ENTRIES] = pn; 
  }
  int physical_addr = (lookup_pt(vm, pn) << 5) + offset;
  uchar ans = vm->buffer[physical_addr];
  // printf("Read: page number %d page fault %d\n", (int) addr, (*(vm->pagefault_num_ptr)));
  return ans; //TODO
}

__device__ void vm_write(VirtualMemory *vm, u32 addr, uchar value) {
  /* Complete vm_write function to write value into data buffer */
  
  int pn = addr >> 5, offset = addr & 31;
  int ret = lookup_pt(vm, pn);
  if(pn % 4 != vm->thread_id && ret != 0x80000000) return;

  if(ret == 0x80000000){ //not found
    (*(vm->pagefault_num_ptr))++;
    int idx = LRU_get(vm); // idx of PT
    u32 replace_page = vm->invert_page_table[idx]; // frame addr
    if(replace_page == 0x80000000) {
      vm->invert_page_table[idx + vm->PAGE_ENTRIES] = pn; 
      vm->invert_page_table[idx] = idx;
    } else {
      u32 pos_in = swap_in_page(vm, pn);  
      for(int i = 0; i < 32; i++) {
        uchar tmp = vm->buffer[(replace_page << 5) + i];
        vm->buffer[(replace_page << 5) + i] = vm->storage[(pos_in << 5) + i];
        vm->storage[(pos_in << 5) + i] = tmp;
      }
      
      swap_out_page(vm, vm->invert_page_table[idx + vm->PAGE_ENTRIES], pos_in);
      // if(pn % 1024 == 0)
      // printf("Write: pn %d fault %d, replace idx %d and pn %d; pos_in %d, vm->buffer %x storage %x val %x\n", 
      // (int) pn, (*(vm->pagefault_num_ptr)), idx, vm->invert_page_table[idx + vm->PAGE_ENTRIES], pos_in, vm->buffer[replace_page << 5], vm->storage[pos_in << 5], value);

      vm->invert_page_table[idx + vm->PAGE_ENTRIES] = pn; 

    }
  }
  int physical_addr = (lookup_pt(vm, pn) << 5) + offset;
  vm->buffer[physical_addr] = value;
  return ; //TODO
}

__device__ void vm_snapshot(VirtualMemory *vm, uchar *results, int offset,
                            int input_size) {
  /* Complete snapshot function togther with vm_read to load elements from data
   * to result buffer */
   for(int i = 0; i < input_size; i++)
    if(((offset + i) >> 5) % 4 == vm->thread_id )
      results[i] = vm_read(vm, offset + i);
}

