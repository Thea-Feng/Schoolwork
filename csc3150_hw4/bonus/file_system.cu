#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ __managed__ u32 gtime = 0;
// __device__ __managed__ FCB root[1];
// __device__ __managed__ FCB *gparent;
__device__ u16 gparent;
__device__ void fs_init(FileSystem *fs, uchar *volume, int SUPERBLOCK_SIZE,
							int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
							int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE, 
							int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS)
{
  // init variables
  fs->volume = volume;

  // init constants
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;
  fs->FCB_SIZE = FCB_SIZE;
  fs->FCB_ENTRIES = FCB_ENTRIES;
  fs->STORAGE_SIZE = VOLUME_SIZE;
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;
  fs->MAX_FILE_NUM = MAX_FILE_NUM;
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;
  fs->VCB = reinterpret_cast<VCB *> (fs->volume);
  fs->FCB = reinterpret_cast<FCB *> (fs->volume + 50);
  fs->SB = reinterpret_cast<uchar *> (fs->volume + fs->FILE_BASE_ADDRESS);
  gparent = 2025;
  fs->current_dir = 2024;
}

__device__ bool cmp(char *A, char *B){
  for(int i = 0; i < 20; i++)
    if(A[i] != B[i]) return 0;
    else if(A[i] == '\0' && B[i] == '\0') return 1;
    return 0;
}
__device__ void cpy(char *A, char *B){
  for(int i = 0; i < 20; i++){
    A[i] = B[i];
    if(B[i] == '\0') return;
  }
}
__device__ int get_lenght(char *A){
  int ret = 0;
  for(int i = 0; i < 20 && A[i] != '\0'; i++, ret++);
  return ret+1;
}
__device__ u32 fs_open(FileSystem *fs, char *s, int op)
{
	/* Implement open operation here */
  // int create_idx = gtime;
  for(int i = 0; i < fs->VCB->tot_file_num; i++){
    if(cmp(fs->FCB[i].file_name, s) && (fs->FCB[i].parent == fs->current_dir))
      return i;
  }
  if(op == G_WRITE){
    //update vcb
    int fp = fs->VCB->tot_file_num++;
    fs->FCB[fp].size = 0;
    fs->FCB[fp].size2 = 0;
    fs->FCB[fp].dir = false;   
    fs->FCB[fp].create = gtime;
    if (fs->current_dir != 2024) 
      fs->FCB[fs->current_dir].modified = gtime;
    fs->FCB[fp].modified = gtime++;
    fs->FCB[fp].st_block = fs->VCB->empty_st++;

    cpy(fs->FCB[fp].file_name, s);
    fs->FCB[fp].parent = fs->current_dir;
    if(fs->current_dir != 2024)
      fs->FCB[fs->current_dir].size += get_lenght(s);
    // fs->FCB_now->parent->size += get_lenght(s);

    return fp;
  }
  printf("Read file DNE\n");
  return 2024;
}


__device__ void fs_read(FileSystem *fs, uchar *output, u32 size, u32 fp)
{
	/* Implement read operation here */
  int st_addr = fs->FCB[fp].st_block * fs->STORAGE_BLOCK_SIZE;
  for(int i = st_addr; i < st_addr + size; i++){
    output[i - st_addr] = fs->SB[i];
  }
}

__device__ u32 fs_write(FileSystem *fs, uchar* input, u32 size, u32 fp)
{
	/* Implement write operation here */
  int pre_size =  (fs->FCB[fp].size + fs->FCB[fp].size2 / 60000);
  int block_num = (size + fs->STORAGE_BLOCK_SIZE  - 1) / fs->STORAGE_BLOCK_SIZE;
  int pre_block_num = pre_size == 0 ? 1 : (pre_size + fs->STORAGE_BLOCK_SIZE  - 1) / fs->STORAGE_BLOCK_SIZE;
  // overwrite, count previous block;
  int change_num = (block_num - pre_block_num);
  // printf("Write: %s %d %d %d\n", fs->FCB[fp].file_name, change_num, size, pre_block_num);

  // move sb
  if(fp < fs->VCB->tot_file_num - 1)
  memcpy(fs->SB + (fs->FCB[fp + 1].st_block + change_num) * fs->STORAGE_BLOCK_SIZE,
  fs->SB + fs->FCB[fp + 1].st_block * fs->STORAGE_BLOCK_SIZE,  (fs->VCB->empty_st - fs->FCB[fp + 1].st_block) * fs->STORAGE_BLOCK_SIZE);

  // update vcb
  fs->VCB->empty_st += change_num;
  //update fcb
  fs->FCB[fp].modified = gtime++;
  fs->FCB[fp].size = size % 60000;
  fs->FCB[fp].size2 = size / 60000;

  for(int i = fp + 1; i < fs->VCB->tot_file_num; i++){
    fs->FCB[i].st_block += change_num;
  }
  //update sb
  int st_addr = fs->FCB[fp].st_block * fs->STORAGE_BLOCK_SIZE;
  for(int i = st_addr; i < st_addr + size; i++){
    fs->SB[i] = input[i - st_addr];
  }
  return fp; // return ?
}
__device__ void fs_gsys(FileSystem *fs, int op){
	/* Implement LS_D and LS_S operation here */
 if(op == LS_D) {
    printf("===sort by modified time===\n");
    for(u16 i = 0; i < fs->VCB->tot_file_num; i++)
      fs->FCB[i].priority = 2024;  
      for(int i = 0; i < fs->VCB->tot_file_num; i++) {
      int idx = -1;
    for(int j = 0; j < fs->VCB->tot_file_num; j++) {
      if(fs->FCB[j].priority == 2024 && (idx == -1 || fs->FCB[idx].modified < fs->FCB[j].modified))
          idx = j;
      }
      fs->FCB[idx].priority = i;
    }

    for(int j = 0; j < fs->VCB->tot_file_num; j++)
      for(int i = 0; i < fs->VCB->tot_file_num; i++) {
        if(fs->FCB[i].priority == j && fs->FCB[i].parent == fs->current_dir){
          if(fs->FCB[i].dir) 
            printf("%s d\n", fs->FCB[i].file_name);
          else printf("%s\n", fs->FCB[i].file_name);
        }
      }
  }
  else if(op == LS_S) {
    printf("===sort by file size===\n");

    // version 1
    for(u16 i = 0; i < fs->VCB->tot_file_num; i++)
      fs->FCB[i].priority = 2024;
    for(int i = 0; i < fs->VCB->tot_file_num; i++) {
      int idx = -1;
      for(int j = 0; j < fs->VCB->tot_file_num; j++) {
        int idx_size = fs->FCB[idx].size + fs->FCB[idx].size2 * 60000;
        int j_size = fs->FCB[j].size + fs->FCB[j].size2 * 60000;
        if(fs->FCB[j].priority == 2024 && (idx == -1 || idx_size < j_size || 
        (idx_size == j_size && fs->FCB[idx].create > fs->FCB[j].create)))
          idx = j;
      }
      fs->FCB[idx].priority = i;
    }
      
    for(int j = 0; j < fs->VCB->tot_file_num; j++)
      for(int i = 0; i < fs->VCB->tot_file_num; i++) {
        if(fs->FCB[i].priority == j && fs->FCB[i].parent == fs->current_dir){
          if(fs->FCB[i].dir) printf("%s %u d\n", fs->FCB[i].file_name, fs->FCB[i].size);
          else printf("%s %u\n", fs->FCB[i].file_name, fs->FCB[i].size);
        }
      }
    // dump(fs);
  } 
  else if(op == CD_P) {
    fs->current_dir = fs->FCB[fs->current_dir].parent;
  } else if(op == PWD) {
    char name_string[60];
    name_string[0] = name_string[20] = name_string[40] = '\0';
    int dep = 40;
    int tmp = fs->current_dir;
    while(tmp != 2024) {
      cpy(name_string + dep, fs->FCB[tmp].file_name);
      dep -= 20;
      tmp = fs->FCB[tmp].parent;
    }
    for(int i = 0; i < 60; i += 20){
      if(name_string[i] != '\0') {
        char sub_name[20];
        cpy(sub_name, name_string + i);
        printf("/%s", sub_name);
      }
    }  
    if(name_string[0] == '\0' && name_string[20] == '\0' && name_string[40] == '\0') printf("/");                                                                                                                                                                                                                                                                                                   
    printf("\n");
  }
  else 
    printf("Wrong operation\n");
  return ;
}

__device__ void fs_gsys(FileSystem *fs, int op, char *s)
{
	/* Implement rm operation here */
  if(op == RM) {
    bool in_dir = false;
    if(gparent == 2025) {gparent = fs->current_dir; in_dir = true;}
    for(int i = 0; i < fs->VCB->tot_file_num; i++)
      if(cmp(fs->FCB[i].file_name, s) && (fs->FCB[i].parent == gparent)){
          int st_block = fs->FCB[i].st_block;
          int size = (fs->FCB[i].size + fs->FCB[i].size2 / 60000);
          int block_num = fs->FCB[i].size == 0 ? 1 : (fs->FCB[i].size + fs->STORAGE_BLOCK_SIZE - 1) / fs->STORAGE_BLOCK_SIZE;
          // dump(fs);
          if(gparent != 2024) {fs->FCB[gparent].size -= get_lenght(fs->FCB[i].file_name); fs->FCB[gparent].modified = gtime++;}
          fs->VCB->tot_file_num--;
          fs->VCB->empty_st -= block_num;
          for(int j = i; j < fs->VCB->tot_file_num; j++){
            fs->FCB[j] = fs->FCB[j + 1];
            fs->FCB[j].st_block -= block_num;
          
          }

          if(i < fs->VCB->tot_file_num)
          memcpy(fs->SB + st_block * fs->STORAGE_BLOCK_SIZE, fs->SB + (st_block + block_num) * 
          fs->STORAGE_BLOCK_SIZE, (fs->VCB->empty_st - st_block) * fs->STORAGE_BLOCK_SIZE);
          else memset(fs->SB + st_block * fs->STORAGE_BLOCK_SIZE, 0, size);
          gparent = 2025;
          return;
        }
      // else {
      //   printf("%s %s %s\n", fs->FCB[i].file_name, fs->FCB[i].parent->file_name, gparent->file_name);
      // } 
    
    gparent = 2025;
    printf("Delete file DNE\n");
    
    
  
  } 
  else if(op == CD) {
    for(int i = 0; i < fs->VCB->tot_file_num; i++){
      if(fs->FCB[i].parent == fs->current_dir && cmp(fs->FCB[i].file_name, s)){
        fs->current_dir = i;
        return;
      }
    }
      printf("No such subdirectory\n");
  }
  else if(op == MKDIR) {
    for(int i = 0; i < fs->VCB->tot_file_num; i++)
      if(cmp(s, fs->FCB[i].file_name) && fs->FCB[i].parent == fs->current_dir) return;

    int fp = fs->VCB->tot_file_num++;
    //update fcb
    fs->FCB[fp].create = gtime;
    if (fs->current_dir != 2024) 
      fs->FCB[fs->current_dir].modified = gtime;
    fs->FCB[fp].modified = gtime++;
    fs->FCB[fp].size = 0;
    fs->FCB[fp].size2 = 0;

    fs->FCB[fp].dir = true;
    fs->FCB[fp].st_block = fs->VCB->empty_st++;
    cpy(fs->FCB[fp].file_name, s);
    fs->FCB[fp].parent = fs->current_dir;
    if(fs->current_dir != 2024)
      fs->FCB[fs->current_dir].size += get_lenght(s);
  } 
  else if(op == RM_RF) {
    if(gparent == 2025) gparent = fs->current_dir;
    for(int i = 0; i < fs->VCB->tot_file_num; i++)
      if((fs->FCB[i].parent == gparent) && cmp(fs->FCB[i].file_name, s)){
         int idx = 0;
        // printf("%d\n", i);
        for(int j = i + 1; j < fs->VCB->tot_file_num; ){
          if(fs->FCB[j].parent == i) {
              // printf("%d\n", j);
              gparent = i;
              if(fs->FCB[j].dir) fs_gsys(fs, RM_RF, fs->FCB[j].file_name);
              else fs_gsys(fs, RM, fs->FCB[j].file_name);
              idx++;
          }
          else j++;
          // if(idx > 10) {printf("Loop!\n"); break;}
        }
        gparent = fs->current_dir;
        fs_gsys(fs, RM, fs->FCB[i].file_name);  
        gparent = 2025;
        return; 
      }
    gparent = 2025;
    return;
      }
}
__device__ void dump(FileSystem *fs){
  printf("--------------------------------------------------\n");
  for(int i = 0; i < fs->VCB->tot_file_num; i++){
    printf("%d %d %d %s %d ", i, fs->FCB[i].st_block, fs->FCB[i].size, fs->FCB[i].file_name,  fs->FCB[i].dir);
    if(fs->FCB[i].parent == 2024) printf("parent: root\n");
    else printf("parent: %s\n", fs->FCB[fs->FCB[i].parent].file_name);
    // if( fs->FCB[i].first_ch != NULL) printf(" child: %s",  fs->FCB[i].first_ch->file_name);
    // if(  fs->FCB[i].peer != NULL ) printf(" peer: %s",  fs->FCB[i].peer->file_name);
    // printf("\n");
  }
  printf("empty_st: %d", fs->VCB->empty_st);
  if(fs->current_dir == 2024) printf(" root\n");
  else printf(" %s\n", fs->FCB[fs->current_dir].file_name);
  printf("--------------------------------------------------\n");

}


