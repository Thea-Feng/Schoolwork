#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>

#ifdef GUI
#include <GL/glut.h>
#include <GL/gl.h>
#include <GL/glu.h>
#endif

#include "./headers/physics.h"
const int max_iter = 5000;

__device__ __managed__ int n, size, ccnt = 0;
__device__ __managed__ bool cont;
int block_size = 40; // cuda thread block size
// int size; // problem size
// int n;

__global__ void initialize(float *data) {
    // TODO: intialize the temperature distribution (in parallelized way)
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        data[i] = wall_temp;
    }
}


__global__ void generate_fire_area(bool *fire_area){
    // TODO: generate the fire area (in parallelized way)
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        fire_area[idx] = 0;
        int i = idx / size, j = idx % size;
        
        float fire1_r2 = fire_size * fire_size;
        int a = i - size / 2;
        int b = j - size / 2;
        int r2 = 0.5 * a * a + 0.8 * b * b - 0.5 * a * b;
        if (r2 < fire1_r2) fire_area[idx] = 1;
       
        float fire2_r2 = (fire_size / 2) * (fire_size / 2);
        a = i - 1 * size / 3;
        b = j - 1 * size / 3;
        r2 = a * a + b * b;
        if (r2 < fire2_r2) fire_area[idx] = 1;
        
    }
}


__global__ void update(float *data, float *new_data) {
    // TODO: update temperature for each point  (in parallelized way)
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    // printf("%d %d %d %d\n", idx, blockDim.x, blockIdx.x, threadIdx.x);
    if (idx < n) {
        int ii = idx / size, jj = idx % size;
        if(ii == 0 || ii == size - 1 || jj == 0 || jj == size - 1) return;
        float up = data[idx - size];
        float down = data[idx + size];
        float left = data[idx - 1];
        float right = data[idx + 1];
        float new_val = (up + down + left + right) / 4;
        new_data[idx] = new_val;
    }

}



__global__ void maintain_fire(float *data, bool *fire_area) {
    // TODO: maintain the temperature of the fire (in parallelized way)

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {  
        if(fire_area[i]) data[i] = fire_temp;
        __syncthreads();
    }
}


__global__ void check_continue(float *data, float *new_data) {
    // TODO: determine if terminated (in parallelized way) you may need to define extra __device__ and __global__ functions
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {  
        if(abs(data[i] - new_data[i]) > threshold) {
            cont = true;
            // printf("%d\n", cont);

        }
    }
    // printf("%d\n",cont);
     __syncthreads();
}


#ifdef GUI
__global__ void data2pixels(float *data, GLubyte* pixels){
    // TODO: convert rawdata (large, size^2) to pixels (small, resolution^2) for faster rendering speed (in parallelized way)
    float factor_data_pixel = (float) size / resolution;
    float factor_temp_color = (float) 255 / fire_temp;
    int idx_pixel = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx_pixel < resolution * resolution){
        int x = idx_pixel / resolution, y = idx_pixel % resolution;
        int idx_pixel = idx_pixel * 3;
        int x_raw = x * factor_data_pixel;
        int y_raw = y * factor_data_pixel;
        int idx_raw = y_raw * size + x_raw;
        float temp = data[idx_raw];
        int color =  ((int) temp / 5 * 5) * factor_temp_color;
        pixels[idx_pixel] = color;
        pixels[idx_pixel + 1] = 255 - color;
        pixels[idx_pixel + 2] = 255 - color;
    }
}


void plot(GLubyte* pixels){
    // visualize temprature distribution
    #ifdef GUI
    glClear(GL_COLOR_BUFFER_BIT);
    glDrawPixels(resolution, resolution, GL_RGB, GL_UNSIGNED_BYTE, pixels);
    glutSwapBuffers();
    #endif
}
#endif

void master() {
    float *data_odd;
    float *data_even;
    bool *fire_area;
    float *final_data;
    final_data = new float[size * size];
    hipMalloc(&data_odd, size * size * sizeof(float));
    hipMalloc(&data_even, size * size * sizeof(float));
    hipMalloc(&fire_area, size * size * sizeof(bool));

    #ifdef GUI
    GLubyte *pixels;
    GLubyte *host_pixels;
    host_pixels = new GLubyte[resolution * resolution * 3];
    hipMalloc(&pixels, resolution * resolution * 3 * sizeof(GLubyte));
    #endif

    int n_block_size = size * size / block_size + 1;
    int n_block_resolution = resolution * resolution / block_size + 1;

    initialize<<<n_block_size, block_size>>>(data_odd);
    generate_fire_area<<<n_block_size, block_size>>>(fire_area);
    
    int count = 1;
    bool cont_user = true; // if should continue
    double total_time = 0;
    while (cont_user && count < max_iter){
        std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
        cont_user = false;
        hipMemcpy(&cont, &cont_user, sizeof(bool), hipMemcpyHostToDevice);
        // TODO: modify the following lines to fit your need.
        if (count % 2 == 1) {
            update<<<n_block_size, block_size>>>(data_odd, data_even);
            maintain_fire<<<n_block_size, block_size>>>(data_even, fire_area);
            // cont = check_continue(data_odd, data_even);
        } else {
            update<<<n_block_size, block_size>>>(data_even, data_odd);
            maintain_fire<<<n_block_size, block_size>>>(data_odd, fire_area);
        }

        check_continue<<<n_block_size, block_size>>>(data_odd, data_even);
        hipMemcpy(&cont_user, &cont, sizeof(bool), hipMemcpyDeviceToHost);
        
        std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
        double this_time = std::chrono::duration<double>(t2 - t1).count();
        total_time += this_time;
        printf("Iteration %d, elapsed time: %.6f\n", count, this_time);
        count++;
        
        #ifdef GUI
        if (count % 2 == 1) {
            data2pixels<<<n_block_resolution, block_size>>>(data_even, pixels);
        } else {
            data2pixels<<<n_block_resolution, block_size>>>(data_odd, pixels);
        }
        hipMemcpy(host_pixels, pixels, resolution * resolution * 3 * sizeof(GLubyte), hipMemcpyDeviceToHost);
        plot(host_pixels);
        #endif
    }
    
    printf("Converge after %d iterations, elapsed time: %.6f, average computation time: %.6f\n", count-1, total_time, (double) total_time / (count-1));


    hipFree(data_odd);
    hipFree(data_even);
    hipFree(fire_area);

    #ifdef GUI
    hipFree(pixels);
    delete[] host_pixels;
    #endif
    
}


int main(int argc, char *argv[]){
    
    size = atoi(argv[1]);
    n = size * size;
    #ifdef GUI
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
    glutInitWindowPosition(0, 0);
    glutInitWindowSize(resolution, resolution);
    glutCreateWindow("Heat Distribution Simulation Sequential Implementation");
    gluOrtho2D(0, resolution, 0, resolution);
    #endif

    master();

    printf("Student ID: 120090266\n"); // replace it with your student id
    printf("Name: Feng Yutong\n"); // replace it with your name
    printf("Assignment 4: Heat Distribution CUDA Implementation\n");
    return 0;

}


